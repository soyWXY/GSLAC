
#include <hip/hip_runtime.h>
// Device code
extern "C" __global__ void MatMul_kernel(const float *A, const float *B,
                                         float *C, int N, int M) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    float temp = 0.0f;
    if (row < N && col < N) {
        for (int i = 0; i < N; i++) {
            temp += A[row * N + i] * B[i * N + col];
        }
    }
    C[row * N + col] = temp;
}